#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>
#include <math.h>
#include <unistd.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <ctime>
#include <sys/time.h>
#include <omp.h>
#include "readWeights30.h"//to read the weights
#include "deviceFunctions30.h"//contains device functions like matmul,add
using namespace std;
inline int _ConvertSMVer2Cores(int major, int minor)
{
    // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
    typedef struct
    {
        int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] =
    {
        { 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
        { 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
        { 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
        { 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
        { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
        { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
        { 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
        { 0x32, 192}, // Kepler Generation (SM 3.2) GK10x class
        { 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
        { 0x37, 192}, // Kepler Generation (SM 3.7) GK21x class
        { 0x50, 128}, // Maxwell Generation (SM 5.0) GM10x class
        {   -1, -1 }
    };
            int index = 0;

    while (nGpuArchCoresPerSM[index].SM != -1)
    {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
        {
            return nGpuArchCoresPerSM[index].Cores;
        }

        index++;
    }

    // If we don't find the values, we default use the previous one to run properly
    printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[index-1].Cores);
    return nGpuArchCoresPerSM[index-1].Cores;
}




#define CUDA_RT_CALL(call)                                                                  \
    {                                                                                       \
        hipError_t cudaStatus = call;                                                      \
        if (hipSuccess != cudaStatus)                                                      \
            fprintf(stderr,                                                                 \
                    "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "              \
                    "with "                                                                 \
                    "%s (%d).\n",                                                           \
                    #call, __LINE__, __FILE__, hipGetErrorString(cudaStatus), cudaStatus); \
}
__global__ void testKernel() {
	int tid = blockIdx.x*blockDim.x+threadIdx.x;
	printf(" FROM TEST KERNEL %d\n",tid);
         printf("TID IS %d\n", tid);
}
__global__ void predictKernel(double *X,double *W_i,double *W_f,double *W_c,double *W_o,double *U_i,double *U_f,double *U_c,double *U_o,double *b_i,double *b_f,double *b_c,double *b_o,double *w,double *b,double *result,double *loop_count)//cuda kernel
        {
                // Get our global thread ID
                int tid = blockIdx.x*blockDim.x+threadIdx.x;
                //if(tid==31908)
                //printf("Done");
                loop_count[0]=0;
                double x[30][3];//input to lstm,50 timestamps
                double *c_t,*h_t,*i_t,*C_t,*f_t,*o_t;
                double H[30][60];//storing the output of each timestamp(50 timestamps, each output of size 50)
                double input[60],output[12];//input & output of dense layer
                double pd1[12],pd2[12];//probabbility density for upper and lower window resp.
                int i,j;
                double sum,res;
                if ((tid>29&&tid<429887-30))
                {
                        //create upper window
                        #pragma unroll
                        for(i=29;i>=0;i--)//i :timestamp from 0-49
                        {
                                x[i][0]=*(X+(tid-(29-i))*3+0);
                                x[i][1]=*(X+(tid-(29-i))*3+1);
                                x[i][2]=*(X+(tid-(29-i))*3+2);
                                loop_count[0]++;
                        }
                        //prediction  for upper window
                        #pragma unroll
                        for(i=0;i<30;i++)//i: timestamp(t)
                        {
                                if(i==0)
                                {
                                        i_t=sigmoid(add(matmul1(W_i,x[i]),b_i));
                                        C_t=tan(add(matmul1(W_c,x[i]),b_c));
                                        f_t=sigmoid(add(matmul1(W_f,x[i]),b_f));
                                        c_t=mult(i_t,C_t);
                                        o_t=sigmoid(add(matmul1(W_o,x[i]),b_o));
                                        h_t=mult(o_t,tan(c_t));

                                        #pragma unroll
                                        for(j=0;j<30;j++)
					 {
                                        H[i][j]=h_t[j];
                                        loop_count[0]++;
                                        }
                                }//if
                                else
                                {
                                        i_t=sigmoid(add(add(matmul1(W_i,x[i]),matmul2(U_i,h_t)),b_i));
                                        C_t=tan(add(add(matmul1(W_c,x[i]),matmul2(U_c,h_t)),b_c));
                                        f_t=sigmoid(add(add(matmul1(W_f,x[i]),matmul2(U_f,h_t)),b_f));
                                        c_t=add(mult(i_t,C_t),mult(f_t,c_t));
                                        o_t=sigmoid(add(add(matmul1(W_o,x[i]),matmul2(U_o,h_t)),b_o));
                                        h_t=mult(o_t,tan(c_t));
                                        #pragma unroll
                                        for(j=0;j<30;j++)
                                        {
                                                H[i][j]=h_t[j];
                                                loop_count[0]++;
                                        }
                                }//else
                        }
                        //backward pass
                        #pragma unroll
                         for(i=29;i>=0;i--)//i :timestamp from 0-49
                        {
                        x[29-i][0]=*(X+(tid-(29-i))*3+0);
                        x[29-i][1]=*(X+(tid-(29-i))*3+1);
                        x[29-i][2]=*(X+(tid-(29-i))*3+2);
                        loop_count[0]++;
                        }
                        #pragma unroll
                        for(i=0;i<30;i++)//i: timestamp(t)
                        {
                                if(i==0)
                                {
                                        i_t=sigmoid(add(matmul1(W_i,x[i]),b_i));
                                        C_t=tan(add(matmul1(W_c,x[i]),b_c));
                                        f_t=sigmoid(add(matmul1(W_f,x[i]),b_f));
                                        c_t=mult(i_t,C_t);
                                        o_t=sigmoid(add(matmul1(W_o,x[i]),b_o));
                                        h_t=mult(o_t,tan(c_t));
					#pragma unroll
                                        for(j=0;j<30;j++)
                                        {
                                        H[i][30+j]=h_t[j];
                                        loop_count[0]++;
                                        }
                                }//if
                                else
                                {
                                        i_t=sigmoid(add(add(matmul1(W_i,x[i]),matmul2(U_i,h_t)),b_i));
                                        C_t=tan(add(add(matmul1(W_c,x[i]),matmul2(U_c,h_t)),b_c));
                                        f_t=sigmoid(add(add(matmul1(W_f,x[i]),matmul2(U_f,h_t)),b_f));
                                        c_t=add(mult(i_t,C_t),mult(f_t,c_t));
                                        o_t=sigmoid(add(add(matmul1(W_o,x[i]),matmul2(U_o,h_t)),b_o));
                                        h_t=mult(o_t,tan(c_t));
                                        #pragma unroll
                                        for(j=0;j<30;j++)
                                        {
                                                H[i][30+j]=h_t[j];
                                                loop_count[0]++;
                                        }
                                }//else
                        }

                        //Mean Pooling
                        #pragma unroll
                        for(j=0;j<60;j++)
                        {
                                sum=0;
                                #pragma unroll
                                for(i=0;i<30;i++)
                                {
                                        sum+=H[i][j];
                                        loop_count[0]++;
                                }
                                input[j]=sum/(30.0);
                        }
                        //Dense Layer
                        sum=0;
			 #pragma unroll
                        for(i=0;i<12;i++)
                        {
                                output[i]=b[i];
                                #pragma unroll
                                for(j=0;j<60;j++)
                                {
                                        output[i]+=(input[j]*(*(w+j*12+i)));
                                        loop_count[0]++;
                                }
                                sum+=exp(output[i]);
                        }
                        #pragma unroll
                        for(i=0;i<12;i++)//prob density for upper window
                        {
                                pd1[i]=exp(output[i])/sum;
                                loop_count[0]++;
                        }
                        //create lower window
                        #pragma unroll
                        for(i=0;i<30;i++)//i :timestamp from 0-49
                        {
                                x[i][0]=*(X+(tid+i)*3+0);
                                x[i][1]=*(X+(tid+i)*3+1);
                                x[i][2]=*(X+(tid+i)*3+2);
                                loop_count[0]++;
                        }
                        //prediction  for lower window
                        #pragma unroll
                        for(i=0;i<30;i++)//i: timestamp(t)
                        {

                                if(i==0)
                                {
                                        i_t=sigmoid(add(matmul1(W_i,x[i]),b_i));
                                        C_t=tan(add(matmul1(W_c,x[i]),b_c));
                                        f_t=sigmoid(add(matmul1(W_f,x[i]),b_f));
                                        c_t=mult(i_t,C_t);
                                        o_t=sigmoid(add(matmul1(W_o,x[i]),b_o));
                                        h_t=mult(o_t,tan(c_t));
                                        #pragma unroll
					for(j=0;j<30;j++)
                                        {
                                                H[i][j]=h_t[j];
                                                loop_count[0]++;
                                        }
                                }//if
                                else
                                {
                                        i_t=sigmoid(add(add(matmul1(W_i,x[i]),matmul2(U_i,h_t)),b_i));
                                        C_t=tan(add(add(matmul1(W_c,x[i]),matmul2(U_c,h_t)),b_c));
                                        f_t=sigmoid(add(add(matmul1(W_f,x[i]),matmul2(U_f,h_t)),b_f));
                                        c_t=add(mult(i_t,C_t),mult(f_t,c_t));
                                        o_t=sigmoid(add(add(matmul1(W_o,x[i]),matmul2(U_o,h_t)),b_o));
                                        h_t=mult(o_t,tan(c_t));
                                        #pragma unroll
                                        for(j=0;j<30;j++)
                                        {
                                                H[i][j]=h_t[j];
                                                loop_count[0]++;
                                        }
                                }//else
                        }
                //Backward pass
                #pragma unroll
                for(i=0;i<30;i++)//i :timestamp from 0-49
                {
                        x[29-i][0]=*(X+(tid+i)*3+0);
                        x[29-i][1]=*(X+(tid+i)*3+1);
                        x[29-i][2]=*(X+(tid+i)*3+2);
                        loop_count[0]++;
                }
                //prediction  for lower window
                #pragma unroll
                for(i=0;i<30;i++)//i: timestamp(t)
                {
                        if(i==0)
                        {
                                i_t=sigmoid(add(matmul1(W_i,x[i]),b_i));
                                C_t=tan(add(matmul1(W_c,x[i]),b_c));
                                f_t=sigmoid(add(matmul1(W_f,x[i]),b_f));
                                c_t=mult(i_t,C_t);
				o_t=sigmoid(add(matmul1(W_o,x[i]),b_o));
                                h_t=mult(o_t,tan(c_t));
                                #pragma unroll
                                for(j=0;j<30;j++)
                                {
                                        H[i][30+j]=h_t[j];
                                        loop_count[0]++;
                                }
                        }//if
                        else
                        {
                                i_t=sigmoid(add(add(matmul1(W_i,x[i]),matmul2(U_i,h_t)),b_i));
                                C_t=tan(add(add(matmul1(W_c,x[i]),matmul2(U_c,h_t)),b_c));
                                f_t=sigmoid(add(add(matmul1(W_f,x[i]),matmul2(U_f,h_t)),b_f));
                                c_t=add(mult(i_t,C_t),mult(f_t,c_t));
                                o_t=sigmoid(add(add(matmul1(W_o,x[i]),matmul2(U_o,h_t)),b_o));
                                h_t=mult(o_t,tan(c_t));
                                #pragma unroll
                                for(j=0;j<30;j++)
                                {
                                        H[i][30+j]=h_t[j];
                                        loop_count[0]++;
                                }
                        }//else
                }
                        //Mean Pooling
                        #pragma unroll
                        for(j=0;j<60;j++)
                        {
                                sum=0;
                                #pragma unroll
                                for(i=0;i<30;i++)
                                {
                                        sum+=H[i][j];
                                        loop_count[0]++;
                                }
                                input[j]=sum/(30.0);
                        }
                        //Dense Layer
                        sum=0;
                        #pragma unroll
			for(i=0;i<12;i++)
                        {
                                output[i]=b[i];
                                #pragma unroll
                                for(j=0;j<60;j++)
                                {
                                        output[i]+=(input[j]*(*(w+j*12+i)));
                                        loop_count[0]++;
                                }
                                sum+=exp(output[i]);
                        }
                        #pragma unroll
                        for(i=0;i<12;i++)//prob density for upper window
                        {
                                pd2[i]=exp(output[i])/sum;
                                loop_count[0]++;
                        }

                        res=0;
                        #pragma unroll
                        for(i=0;i<12;i++)
                        {
                                res+=(pd1[i]*pd2[i]);
                                loop_count[0]++;
                        }
                        *(result+tid)=res;
                }//if tid
        }// kernel loop
int main()
{
        double *X=(double *)malloc(1719551 * 3 * sizeof(double));//dataset
	double *W_i=(double *)malloc(30*3*sizeof(double));
	double *W_f=(double *)malloc(30*3*sizeof(double));
	double *W_c=(double *)malloc(30*3*sizeof(double));
	double *W_o=(double *)malloc(30*3*sizeof(double));
	double *U_i=(double *)malloc(30*30*sizeof(double));
	double *U_f=(double *)malloc(30*30*sizeof(double));
	double *U_c=(double *)malloc(30*30*sizeof(double));
	double *U_o=(double *)malloc(30*30*sizeof(double));
	double *b_i=(double *)malloc(30*sizeof(double));
	double *b_f=(double *)malloc(30*sizeof(double));
	double *b_c=(double *)malloc(30*sizeof(double));
	double *b_o=(double *)malloc(30*sizeof(double));
	double *w=(double *)malloc(60*12*sizeof(double));
	double *b=(double *)malloc(12*sizeof(double));
	readWeights(X,W_i,W_f,W_c,W_o,U_i,U_f,U_c,U_o,b_i,b_f,b_c,b_o,w,b);//read the weights from file(readWeights.h)
	double *W_i_gpu,*W_f_gpu,*W_c_gpu,*W_o_gpu,*U_i_gpu,*U_f_gpu,*U_c_gpu,*U_o_gpu,*b_i_gpu,*b_f_gpu,*b_c_gpu,*b_o_gpu,*w_gpu,*b_gpu;//device vector
	//Splitting the dataset into four parts for each device
	size_t bytes1=429887*3*sizeof(double);//size in bytes of the vector to be sent to gpu
	size_t bytes2=30*3*sizeof(double);
	size_t bytes3=30*30*sizeof(double);
	size_t bytes4=30*sizeof(double);
	size_t bytes5=60*12*sizeof(double);
	size_t bytes6=12*sizeof(double);
	size_t bytes7=429887*sizeof(double);

	omp_set_num_threads(4);
	int tid=0;
	#pragma omp parallel private(tid, W_i_gpu, W_f_gpu, W_c_gpu, W_o_gpu, U_i_gpu, U_f_gpu, U_c_gpu, U_o_gpu, b_i_gpu, b_f_gpu, b_c_gpu, b_o_gpu, w_gpu, b_gpu)
	{
		tid = omp_get_thread_num();
		hipSetDevice(tid);
		printf("CPU TID IS %d\n",tid);
	

		// Allocate memory for each vector on GPU
		hipMalloc(&W_i_gpu,bytes2);
		hipMalloc(&W_f_gpu,bytes2);
		hipMalloc(&W_c_gpu,bytes2);
		hipMalloc(&W_o_gpu,bytes2);
		hipMalloc(&U_i_gpu,bytes3);
		hipMalloc(&U_f_gpu,bytes3);
		hipMalloc(&U_c_gpu,bytes3);
		hipMalloc(&U_o_gpu,bytes3);
		hipMalloc(&b_i_gpu,bytes4);
		hipMalloc(&b_f_gpu,bytes4);
		hipMalloc(&b_c_gpu,bytes4);
		hipMalloc(&b_o_gpu,bytes4);
		hipMalloc(&w_gpu,bytes5);
		hipMalloc(&b_gpu,bytes6);

		hipMemcpy(W_i_gpu,W_i,bytes2,hipMemcpyHostToDevice);
		hipMemcpy(W_f_gpu,W_f,bytes2,hipMemcpyHostToDevice);
		hipMemcpy(W_c_gpu,W_c,bytes2,hipMemcpyHostToDevice);
		hipMemcpy(W_o_gpu,W_o,bytes2,hipMemcpyHostToDevice);
		hipMemcpy(U_i_gpu,U_i,bytes3,hipMemcpyHostToDevice);
		hipMemcpy(U_f_gpu,U_f,bytes3,hipMemcpyHostToDevice);
		hipMemcpy(U_c_gpu,U_c,bytes3,hipMemcpyHostToDevice);
		hipMemcpy(U_o_gpu,U_o,bytes3,hipMemcpyHostToDevice);
		hipMemcpy(b_i_gpu,b_i,bytes4,hipMemcpyHostToDevice);
		hipMemcpy(b_f_gpu,b_f,bytes4,hipMemcpyHostToDevice);
		hipMemcpy(b_c_gpu,b_c,bytes4,hipMemcpyHostToDevice);
		hipMemcpy(b_o_gpu,b_o,bytes4,hipMemcpyHostToDevice);
		hipMemcpy(w_gpu,w,bytes5,hipMemcpyHostToDevice);
		hipMemcpy(b_gpu,b,bytes6,hipMemcpyHostToDevice);

		int blockSize, gridSize;
		// Number of threads in each thread block
		blockSize = 1024;
		// Number of thread blocks in grid
		gridSize = (int)ceil((float)429887/blockSize);
		// Execute the kernel
		if(tid == 0) {
			int i,j;
			j=0;
			double *X1_gpu, *result1_gpu,*loop_count_gpu;
			double *X1=(double *)malloc(429887*3*sizeof(double));
			for(i=0;i<=429886;i++)
			{
				*(X1+j*3+0)=*(X+i*3+0);
				*(X1+j*3+1)=*(X+i*3+1);
				*(X1+j*3+2)=*(X+i*3+2);
				j++;
			}
			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			double fs_t, fe_t, ft_t;
       			struct timeval t;
		        int  cudaCores, smCount, totalThreads;
        		double f_avg;
			hipMalloc(&X1_gpu, bytes1);
			hipMalloc(&result1_gpu,bytes7);
			hipMalloc(&loop_count_gpu,1*sizeof(double));
			hipMemcpy(X1_gpu,X1,bytes1,hipMemcpyHostToDevice);
			hipDeviceProp_t devProp;
        		hipGetDeviceProperties(&devProp, tid);
        		smCount = devProp.multiProcessorCount;
        		cudaCores = _ConvertSMVer2Cores(devProp.major, devProp.minor);
        		totalThreads=429887-60;
        		gettimeofday(&t, NULL);
        		fs_t = t.tv_sec+(t.tv_usec/1000000.0);

			hipEventRecord(start);
			predictKernel<<<gridSize, blockSize>>>(X1_gpu,W_i_gpu,W_f_gpu,W_c_gpu,W_o_gpu,U_i_gpu,U_f_gpu,U_c_gpu,U_o_gpu,b_i_gpu,b_f_gpu,b_c_gpu,b_o_gpu,w_gpu,b_gpu,result1_gpu,loop_count_gpu);
			hipEventRecord(stop);

			hipDeviceSynchronize();
        		gettimeofday(&t, NULL);
        		fe_t = t.tv_sec+(t.tv_usec/1000000.0);
        		ft_t = fe_t - fs_t;
			double *loop_count=(double *)malloc(1*sizeof(double));
                        hipMemcpy(loop_count,loop_count_gpu,1*sizeof(double),hipMemcpyDeviceToHost);

        		cout<<loop_count[0]<<' '<<smCount<<' '<<cudaCores<<' '<<totalThreads<<'\n';
        		f_avg += (loop_count[0]*smCount*cudaCores*totalThreads*10)/(ft_t*1000000000);

			CUDA_RT_CALL(hipGetLastError());
			hipDeviceSynchronize();
			double *result1=(double *)malloc(429887*sizeof(double));
			hipMemcpy(result1,result1_gpu,bytes7,hipMemcpyDeviceToHost);
			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			cout<<"Time:"<<'\n';
		        cout<<(float)(milliseconds/1000)<<'\n';
			 printf("Number of FLOPs: %lf G-FLOPs\n", (f_avg));
			hipFree(result1_gpu);
			hipFree(X1_gpu);
			for(int z=31908;z<=31968;z++)
				cout<<result1[z]<<' ';
			cout<<'\n';
	
		}
		if (tid == 1) {
			int i,j;
			j=0;
			double *X2_gpu, *result2_gpu,*loop_count_gpu;
			double *X2=(double *)malloc(429887*3*sizeof(double));
			j=0;
			for(i=429887;i<=859773;i++)
        		{
				*(X2+j*3+0)=*(X+i*3+0);
			        *(X2+j*3+1)=*(X+i*3+1);
			        *(X2+j*3+2)=*(X+i*3+2);
			        j++;
			}
			hipMalloc(&X2_gpu, bytes1);
			hipMalloc(&result2_gpu,bytes7);
			hipMalloc(&loop_count_gpu,1*sizeof(double));
			hipMemcpy(X2_gpu,X2,bytes1,hipMemcpyHostToDevice);
                        predictKernel<<<gridSize, blockSize>>>(X2_gpu,W_i_gpu,W_f_gpu,W_c_gpu,W_o_gpu,U_i_gpu,U_f_gpu,U_c_gpu,U_o_gpu,b_i_gpu,b_f_gpu,b_c_gpu,b_o_gpu,w_gpu,b_gpu,result2_gpu,loop_count_gpu);
			CUDA_RT_CALL(hipGetLastError());
			hipDeviceSynchronize();
			double *result2=(double *)malloc(429887*sizeof(double));
			hipMemcpy(result2,result2_gpu,bytes7,hipMemcpyDeviceToHost);
			hipFree(result2_gpu);
			hipFree(X2_gpu);

			for(int z=31908;z<=31968;z++)
				cout<<result2[z]<<' ';
			cout<<'\n';
	
		}
		if (tid == 2) {
			int i,j;
			j=0;
			double *X3_gpu, *result3_gpu,*loop_count_gpu;
			double *X3=(double *)malloc(429887*3*sizeof(double));
		        for(i=859774;i<=1289660;i++)
		        {
				*(X3+j*3+0)=*(X+i*3+0);
			        *(X3+j*3+1)=*(X+i*3+1);
			        *(X3+j*3+2)=*(X+i*3+2);
				j++;
        		}
			hipMalloc(&X3_gpu, bytes1);
			hipMalloc(&result3_gpu,bytes7);
			hipMalloc(&loop_count_gpu,1*sizeof(double));
			hipMemcpy(X3_gpu,X3,bytes1,hipMemcpyHostToDevice);
                        predictKernel<<<gridSize, blockSize>>>(X3_gpu,W_i_gpu,W_f_gpu,W_c_gpu,W_o_gpu,U_i_gpu,U_f_gpu,U_c_gpu,U_o_gpu,b_i_gpu,b_f_gpu,b_c_gpu,b_o_gpu,w_gpu,b_gpu,result3_gpu,loop_count_gpu);
			CUDA_RT_CALL(hipGetLastError());
			hipDeviceSynchronize();
			double *result3=(double *)malloc(429887*sizeof(double));
			hipMemcpy(result3,result3_gpu,bytes7,hipMemcpyDeviceToHost);
			hipFree(result3_gpu);
			hipFree(X3_gpu);

			for(int z=31908;z<=31968;z++)
				cout<<result3[z]<<' ';
			cout<<'\n';
	
		}
		if(tid == 3) {
			int i,j;
			j=0;
			double *X4_gpu, *result4_gpu,*loop_count_gpu;
			double *X4=(double *)malloc(429887*3*sizeof(double));
		        for(i=1289661;i<=1719547;i++)
        		{
				*(X4+j*3+0)=*(X+i*3+0);
			        *(X4+j*3+1)=*(X+i*3+1);
			        *(X4+j*3+2)=*(X+i*3+2);
        			j++;
			}
			hipMalloc(&X4_gpu, bytes1);
			hipMalloc(&result4_gpu,bytes7);
			hipMalloc(&loop_count_gpu,1*sizeof(double));
			hipMemcpy(X4_gpu,X4,bytes1,hipMemcpyHostToDevice);
                        predictKernel<<<gridSize, blockSize>>>(X4_gpu,W_i_gpu,W_f_gpu,W_c_gpu,W_o_gpu,U_i_gpu,U_f_gpu,U_c_gpu,U_o_gpu,b_i_gpu,b_f_gpu,b_c_gpu,b_o_gpu,w_gpu,b_gpu,result4_gpu,loop_count_gpu);
			CUDA_RT_CALL(hipGetLastError());
			hipDeviceSynchronize();
			double *result4=(double *)malloc(429887*sizeof(double));
			hipMemcpy(result4,result4_gpu,bytes7,hipMemcpyDeviceToHost);
			hipFree(result4_gpu);
			hipFree(X4_gpu);

			for(int z=31908;z<=31968;z++)
				cout<<result4[z]<<' ';
			cout<<'\n';
	
		}
		hipFree(W_i_gpu);
		hipFree(W_f_gpu);
		hipFree(W_c_gpu);
		hipFree(W_o_gpu);
		hipFree(U_i_gpu);
		hipFree(U_f_gpu);
		hipFree(U_c_gpu);
		hipFree(U_o_gpu);
		hipFree(b_i_gpu);
		hipFree(b_f_gpu);
		hipFree(b_c_gpu);
		hipFree(b_o_gpu);
		hipFree(w_gpu);
		hipFree(b_gpu);

	}

        return 0;
}
